#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "ImageHandler.h"
#include "ImageModel.h"
#include "helper.h"

#define BLOCK_SIZE 32 // ideal blocksize for preformance: 32 x 32 = 1024 => block core count
enum Filter
{
	BoxBlur = 0,
	Sharpen = 1
};

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort)
			exit(code);
	}
}
typedef unsigned char byte_t;
__global__ void convolution(float* pixelMap, float* filter, float* resultMap, int width, int height, int components, const int FILTER_SIZE) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	const int filterRadius = FILTER_SIZE / 2;
	if (i >= width || j >= height)
		return;
	for (int z = 0; z < components; z++) {// iterate thru colors
		float sum = 0.0;
		for (int x = -filterRadius; x <= filterRadius; x++) // iterate thru filter rows
			for (int y = -filterRadius; y <= filterRadius; y++) // iterate thru filter cols
				sum += (i + x >= width || i + x < 0 || y + j >= height || y + j < 0)
					? 0 // edge ignore solution
					: filter[(x + 1) * FILTER_SIZE + (y + 1)] // filter x pixel[color]
						* pixelMap[((i + x) * width + (j + y)) * components + z];
		resultMap[(i * width + j) * components + z] = sum;
	}
}

int main(char** argv, int argc) {
	float* d_pixelMap, * d_resultMap, * h_resultMap, *** filters;
	char** filter_names;
	int* filter_sizes, filter_count;
	int size;
	//-----------------

	readFilters("filters.txt",&filters,&filter_sizes,&filter_names, &filter_count);
	int pick = showMenu(filter_names, filter_count);
	const int FILTER_SIZE = filter_sizes[pick];

	auto inputImage = importPPM("lena.ppm");
	auto outputImage = Image_new(inputImage->width, inputImage->height, inputImage->channels);
	size = inputImage->width * inputImage->height * inputImage->channels;
	float* flatFilter = flatenFilter(filters[pick], FILTER_SIZE);
	float* d_filter;
	/*
	Declare and allocate host and device memory. <
	Initialize host data. <
	Transfer data from the host to the device. <
	Execute one or more kernels. <
	Transfer results from the device to the host. <
	*/
	// malloc
	gpuErrchk(hipMalloc((void**)&d_filter, sizeof(float) * FILTER_SIZE * FILTER_SIZE));
	gpuErrchk(hipMalloc((void**)&d_pixelMap, sizeof(float) * size));
	gpuErrchk(hipMalloc((void**)&d_resultMap, sizeof(float) * size));
	//---cpy
	gpuErrchk(hipMemcpy(d_pixelMap, inputImage->data, size * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_filter, flatFilter, sizeof(float) * FILTER_SIZE * FILTER_SIZE, hipMemcpyHostToDevice));
	//DO STUFF

	dim3 numberOfBlocks(ceil(inputImage->width) / BLOCK_SIZE, ceil(inputImage->height / BLOCK_SIZE)); // this divides the image to 32x32+/- blocks
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE); // set to 32 x 32 = 1024 This is the maximum thread count per block (best preformance)

	auto start = std::chrono::high_resolution_clock::now();
	convolution <<<numberOfBlocks, threadsPerBlock>>> (d_pixelMap, d_filter, d_resultMap, inputImage->width, inputImage->height, 3, FILTER_SIZE);
	auto end = std::chrono::high_resolution_clock::now();

	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	gpuErrchk(hipPeekAtLastError());
	printf("Success! Took %I64d mqs\n", duration);

	h_resultMap = (float*)malloc(sizeof(float) * inputImage->width * inputImage->height * inputImage->channels);

	gpuErrchk(hipMemcpy(h_resultMap, d_resultMap, size * sizeof(float), hipMemcpyDeviceToHost));

	outputImage->data = h_resultMap;


	exportPPM("output.ppm", outputImage);

	if (shouldRunSequential()) {
		auto seq_outputImage = Image_new(inputImage->width, inputImage->height, inputImage->channels);
		auto seq_start = std::chrono::high_resolution_clock::now();
		seq_outputImage->data = sequencialConvolution(inputImage->data, flatFilter, inputImage->width, inputImage->height, inputImage->channels, filter_sizes[pick]);
		auto seq_end = std::chrono::high_resolution_clock::now();
		auto seq_duration = std::chrono::duration_cast<std::chrono::microseconds>(seq_end - seq_start).count();
		printf("Success! CPU convolution took %I64d mqs\n", seq_duration);
		printf("Speed up of %d times!\n", seq_duration / duration);
		exportPPM("seq.ppm", seq_outputImage);
		Image_delete(seq_outputImage);
		flushStdinSafe();
		getchar();
	}

	char* ext = ".bmp";
	char output[32];
	char base[64] = "magick output.ppm ";
	strcpy(output, filter_names[pick]);
	strcat(output, ext);
	system(strcat(base, output));
	system(output);

	//clean up
	free(flatFilter);
	hipFree(d_filter);
	hipFree(d_resultMap);
	hipFree(d_pixelMap);
	Image_delete(inputImage);
	Image_delete(outputImage);
	return 0;
}